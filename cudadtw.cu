/*  
 * Copyright (C) 2024 Wuhan University of Technology
 * Authors: Wang Zihao <qianlkzf@outlook.com> 
 *  
 * This program is free software: you can redistribute it and/or modify  
 * it under the terms of the GNU General Public License as published by  
 * the Free Software Foundation, either version 3 of the License, or  
 * (at your option) any later version.  
 *  
 * This program is distributed in the hope that it will be useful,  
 * but WITHOUT ANY WARRANTY; without even the implied warranty of  
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the  
 * GNU General Public License for more details.  
 *  
 * You should have received a copy of the GNU General Public License  
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.  
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern __shared__ float array[];

__global__ void calc_dtw (
    unsigned SRC_LEN,
    unsigned TRG_LEN,
    unsigned TRG_COT,
    float *S, 
    float *TT, 
    float *Result)
{
    // blockDim.x = TRG_LEN*TRG_COT
    float* path_h1 = (float*)array;
    float* path_h2 = (float*)&path_h1[blockDim.x];
    float* dist    = (float*)&path_h2[blockDim.x];

    //int blockId = (blockIdx.y*gridDim.x + blockIdx.x);
    //int G_idx   = blockId *blockDim.x +threadIdx.x;
    float *T =     TT + (blockIdx.y*gridDim.x +blockIdx.x) *blockDim.x;
    float *R = Result + (blockIdx.y*gridDim.x +blockIdx.x) *TRG_COT;
    
    float *ex;
    int i,j;
    
    int sub_x = threadIdx.x % TRG_LEN;
    int x_cot = threadIdx.x / TRG_LEN;
    
    // first line speical, do first
    // 1. paralle, first line's every element's dist
    dist[threadIdx.x] = (S[0] -T[threadIdx.x])
                       *(S[0] -T[threadIdx.x]);
    __syncthreads();
    // 2. serie, first line's every element's serie's dist
    if (sub_x == 0){
        path_h1[threadIdx.x] = dist[threadIdx.x];
        for (i=1; i <TRG_LEN; i++) {
            path_h1[i +threadIdx.x] = path_h1[i-1 +threadIdx.x]
                                    +    dist[i   +threadIdx.x];
        }
    }
    __syncthreads();
    
    for (i=1; i <SRC_LEN; i++){ // do circle
        // 1. paralle, calc itself's DISTANCE, for speed follow progress
        //    use memeory to rise speed
        dist[threadIdx.x] = (S[i] -T[threadIdx.x])
                           *(S[i] -T[threadIdx.x]);
        __syncthreads();
        // 2. paralle, get from upper line's "up","left-up"- the min dist
        if (sub_x == 0) // FIRST element speical, add DISTANCE here
            path_h2[threadIdx.x] =  path_h1[threadIdx.x] + dist[threadIdx.x];
        else
            path_h2[threadIdx.x] = min (path_h1[threadIdx.x],
                                        path_h1[threadIdx.x-1]);
        __syncthreads();
        // 2. serie, compare to left(front) element with myself, get the less value
        if (sub_x == 0) // the first element had plused dist, so can use here
            for (j=1; j<TRG_LEN; j++) 
                path_h2[j +threadIdx.x] = min(path_h2[j   +threadIdx.x], 
                                              path_h2[j-1 +threadIdx.x])
                                        + dist[j +threadIdx.x];
        __syncthreads();
        ex      = path_h2;
        path_h2 = path_h1;
        path_h1 = ex;
        __syncthreads();
    }
    // when all done, can return the result
    if (sub_x == 0){
        R[x_cot] = sqrt(ex[threadIdx.x +TRG_LEN -1]);
    }
}    
